#include "hip/hip_runtime.h"
// OpenMP parallel 2D advection solver module
// written for COMP4300/8300 Assignment 2, 2017
// v1.0 28 Apr 

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "serAdvect.h" // advection parameters

static int M, N, Gx, Gy, Bx, By; // local store of problem parameters
static int verbosity;

//sets up parameters above
void initParParams(int M_, int N_, int Gx_, int Gy_, int Bx_, int By_, 
		   int verb) {
  M = M_, N = N_; Gx = Gx_; Gy = Gy_;  Bx = Bx_; By = By_; 
  verbosity = verb;
} //initParParams()

/****************kernel for parallelization PAR2D*******************/
__global__ void updateBoundaryEWpar(int M, int N, double *u, int ldu) {
  int xid = threadIdx.x + blockDim.x * blockIdx.x;
  int yid = threadIdx.y + blockDim.y * blockIdx.y;
  int numy = blockDim.y * gridDim.y; // total threads at y direction
  int numx = blockDim.x * gridDim.x; // total threads at x direction
  int id1D = xid + yid*numx; // the 1D thread id
  for (int i=(id1D+1); i < M+1; i+=numy*numx) { 
    V(u, i, 0)   = V(u, i, N);
    V(u, i, N+1) = V(u, i, 1);
  }
}

__global__ void updateBoundaryNSpar(int N, int M, double *u, int ldu) {
  int xid = threadIdx.x + blockDim.x * blockIdx.x;
  int yid = threadIdx.y + blockDim.y * blockIdx.y;
  int numy = blockDim.y * gridDim.y; // total threads at y direction
  int numx = blockDim.x * gridDim.x; // total threads at x direction
  int id1D = xid + yid*numx; // the 1D thread id
  for (int j=id1D; j < N+2; j+=numx*numy) { 
    V(u, 0, j)   = V(u, M, j);
    V(u, M+1, j) = V(u, 1, j);
  }  
}

__global__ void updateAdvect1par(int M, int N, double *u, int ldu, double *ut, 
			      int ldut, double dt, double sx, double sy) {
  int xid = threadIdx.x + blockDim.x * blockIdx.x;
  int yid = threadIdx.y + blockDim.y * blockIdx.y;
  int numy = blockDim.y * gridDim.y; // total threads at y direction
  int numx = blockDim.x * gridDim.x; // total threads at x direction
  for (int j=xid; j < N+1; j+=numx) 
    for (int i=yid; i < M+1; i+=numy)
      V(ut,i,j) = 0.25*(V(u,i,j) + V(u,i-1,j) + V(u,i,j-1) + V(u,i-1,j-1))
	-0.5*dt*(sy*(V(u,i,j) + V(u,i,j-1) - V(u,i-1,j) - V(u,i-1,j-1)) +
		 sx*(V(u,i,j) + V(u,i-1,j) - V(u,i,j-1) - V(u,i-1,j-1)));
} 

__global__ void updateAdvect2par(int M, int N, double *u, int ldu, double *ut, 
			      int ldut, double dtdx, double dtdy) {
  int xid = threadIdx.x + blockDim.x * blockIdx.x;
  int yid = threadIdx.y + blockDim.y * blockIdx.y;
  int numy = blockDim.y * gridDim.y; // total threads at y direction
  int numx = blockDim.x * gridDim.x; // total threads at x direction
  for (int j=xid; j < N; j+=numx) 
    for (int i=yid; i < M; i+=numy)
      V(u, i, j) +=
	- dtdy * (V(ut,i+1,j+1) + V(ut,i+1,j) - V(ut,i,j) - V(ut,i,j+1))
	- dtdx * (V(ut,i+1,j+1) + V(ut,i,j+1) - V(ut,i,j) - V(ut,i+1,j));
}

// evolve advection over reps timesteps, with (u,ldu) containing the field
// parallel (2D decomposition) variant
void cuda2DAdvect(int reps, double *u, int ldu) {
  int ldut = M+1;
  double *ut;
  HANDLE_ERROR( hipMalloc(&ut, ldut*(N+1)*sizeof(double)) );
  
  dim3 dimG(Gx, Gy);
  dim3 dimB(Bx, By);
  
  for (int r = 0; r < reps; r++) {
    updateBoundaryEWpar <<<dimG,dimB>>> (M, N, u, ldu);
    updateBoundaryNSpar <<<dimG,dimB>>> (N, M, u, ldu);

    double sx = 0.5 * Velx / deltax, sy = 0.5 * Vely / deltay;
    updateAdvect1par <<<dimG,dimB>>> (M, N, &V(u,1,1), ldu, ut, ldut, dt, sx, sy);

    double dtdx = 0.5 * dt / deltax, dtdy = 0.5 * dt / deltay;
    updateAdvect2par <<<dimG,dimB>>> (M, N, &V(u,1,1), ldu, ut, ldut, dtdx, dtdy);
  } //for(r...)
    
  HANDLE_ERROR( hipFree(ut) );
} //cuda2DAdvect()


/****************kernel for parallelization OPT *******************/
__device__ void updateAdvect1Opt(int M, int N, double *u, int ldu, double *ut, 
			      int ldut, double dt, double sx, double sy) {
  int xid = threadIdx.x + blockDim.x * blockIdx.x;
  int yid = threadIdx.y + blockDim.y * blockIdx.y;
  int numy = blockDim.y * gridDim.y; // total threads at y direction
  int numx = blockDim.x * gridDim.x; // total threads at x direction
  for (int j=xid; j < N+1; j+=numx) 
    for (int i=yid; i < M+1; i+=numy)
      V(ut,i,j) = 0.25*(V(u,i,j) + V(u,i-1,j) + V(u,i,j-1) + V(u,i-1,j-1))
	-0.5*dt*(sy*(V(u,i,j) + V(u,i,j-1) - V(u,i-1,j) - V(u,i-1,j-1)) +
		 sx*(V(u,i,j) + V(u,i-1,j) - V(u,i,j-1) - V(u,i-1,j-1)));
} 

__device__ void updateAdvect2Opt(int M, int N, double *u, int ldu, double *ut, 
			      int ldut, double dtdx, double dtdy) {
  int xid = threadIdx.x + blockDim.x * blockIdx.x;
  int yid = threadIdx.y + blockDim.y * blockIdx.y;
  int numy = blockDim.y * gridDim.y; // total threads at y direction
  int numx = blockDim.x * gridDim.x; // total threads at x direction
  for (int j=xid; j < N; j+=numx) 
    for (int i=yid; i < M; i+=numy)
      V(u, i, j) +=
	- dtdy * (V(ut,i+1,j+1) + V(ut,i+1,j) - V(ut,i,j) - V(ut,i,j+1))
	- dtdx * (V(ut,i+1,j+1) + V(ut,i,j+1) - V(ut,i,j) - V(ut,i+1,j));
}

// usiing shared memory.
// 1. cp boundary from global memory to shared memory
// 2. cp center grid from global memory to shared memory
// 3. updateAdvect1
// 4. synchronize
// 5. updateAdvect2
// 6. synchronize
// 7. write cu back to u

__global__ void cudaOptAdvectKernel(int M, int N, double *u, int ldu, double *ut, int ldut, double dt, double sx, double sy,double dtdx, double dtdy) {

  int xid = threadIdx.x + blockDim.x * blockIdx.x;
  int yid = threadIdx.y + blockDim.y * blockIdx.y;
  int numy = blockDim.y * gridDim.y; // total threads at y direction
  int numx = blockDim.x * gridDim.x; // total threads at x direction

  extern __shared__ char shared[];
  double *cu = (double *) shared;
  double *cut = (double *)((M+2)*(N+2)*sizeof(double) + shared);
  int ldcu = M + 2;
  int ldcut = M + 1;

  // cp boundary from global memory to shared memory
  if ( threadIdx.x == 0 ) { // East
    for (int j=xid+1; j < N+1; j+=numx) 
      for (int i=yid+1; i < M+1; i+=numy)
        V(cu, i, j-1)   = V(u, i, j-1);
  }
  if ( threadIdx.x == blockDim.x-1 ) { // West
    for (int j=xid+1; j < N+1; j+=numx) 
      for (int i=yid+1; i < M+1; i+=numy)
        V(cu, i, j+1)   = V(u, i, j+1);
  }
  if ( threadIdx.y == 0 ) { // North
    for (int j=xid+1; j < N+1; j+=numx){ 
      for (int i=yid+1; i < M+1; i+=numy) {
        V(cu, i-1, j)   = V(u, i-1, j);
        if ( threadIdx.x == 0 ) {
          V(cu, i-1, j-1) = V(u,i-1,j-1);
        } else if ( threadIdx.x == blockDim.x - 1 ) {
          V(cu, i-1, j+1) = V(u, i-1, j+1);
        }
      }
    }
  }
  if ( threadIdx.y == blockDim.y-1 ) { // East
    for (int j=xid+1; j < N+1; j+=numx) {
      for (int i=yid+1; i < M+1; i+=numy){
        V(cu, i+1, j)   = V(u, i+1, j);
        if ( threadIdx.x == 0 ) {
          V(cu, i+1, j-1) = V(u,i+1,j-1);
        } else if ( threadIdx.x == blockDim.x - 1 ) {
          V(cu, i+1, j+1) = V(u, i+1, j+1);
        }
      }
    }
  }
  // cp center grid from global memory to shared memory
  // pay attention to +2, it update the west and south boundary of cu, if total threads cannot be divided by M,N
  for (int j=xid+1; j < N+1; j+=numx)  
    for (int i=yid+1; i < M+1; i+=numy)
      V(cu,i,j) = V(u,i,j);
  __syncthreads();
  // updateAdvect1
  updateAdvect1Opt(M, N, &V(cu,1,1), ldcu, cut, ldcut, dt, sx, sy);
  // synchronize
  __syncthreads();
  // updateAdvect2
  updateAdvect2Opt(M, N, &V(cu,1,1), ldcu, cut, ldcut, dtdx, dtdy);
  // synchronize
  __syncthreads();
  // write cu back to u
  for (int j=xid+1; j < N+1; j+=numx) 
    for (int i=yid+1; i < M+1; i+=numy)
      V(u,i,j) = V(cu,i,j);
}

// ... optimized parallel variant
// using shared memory to do optimization
void cudaOptAdvect(int reps, double *u, int ldu, int w) {
  int ldut = M+1;
  double *ut;
  HANDLE_ERROR( hipMalloc(&ut, ldut*(N+1)*sizeof(double)) );
  
  dim3 dimG(Gx, Gy);
  dim3 dimB(Bx, By);
  //int sharedDim = (Bx+2) * (By+2);
  
  for (int r = 0; r < reps; r++) {
    double sx = 0.5 * Velx / deltax, sy = 0.5 * Vely / deltay;
    double dtdx = 0.5 * dt / deltax, dtdy = 0.5 * dt / deltay;

    updateBoundaryEWpar <<<dimG,dimB>>> (M, N, u, ldu);
    updateBoundaryNSpar <<<dimG,dimB>>> (N, M, u, ldu);
    cudaOptAdvectKernel <<<dimG, dimB,((M+2)*(N+2)+(M+1)*(N+1))*sizeof(double)>>> (M, N, u, ldu, ut, ldut, dt, sx, sy, dtdx, dtdy);
  } //for(r...)
    
  HANDLE_ERROR( hipFree(ut) );
} //cudaOptAdvect()
//void cudaOptAdvect(int reps, double *u, int ldu, int w) { }
